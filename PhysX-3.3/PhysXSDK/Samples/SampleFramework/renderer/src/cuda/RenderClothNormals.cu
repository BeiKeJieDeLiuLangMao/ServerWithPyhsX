#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2008-2015, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

#include "PxPhysics.h"
#include "PxVec4.h"
#include "PxVec3.h"
#include "PxVec2.h"

#include "cloth/PxClothTypes.h"

namespace physx
{

// interleaved format must match that used by RendererClothShape
struct Vertex
{
	PxVec3 position;
	PxVec3 normal;
};

namespace
{
	__device__ inline void PxAtomicFloatAdd(float* dest, float x)
	{
#if __CUDA_ARCH__ >= 200
		atomicAdd(dest, x);
#else
		union bits { float f; unsigned int i; };
		bits oldVal, newVal;

		do
		{
			// emulate atomic float add on 1.1 arch
			oldVal.f = *dest;
			newVal.f = oldVal.f + x;
		}
		while (atomicCAS((unsigned int*)dest, oldVal.i, newVal.i) != oldVal.i);
#endif
	}


	__device__ void PxAtomicVec3Add(PxVec3& dest, PxVec3 inc)
	{
		PxAtomicFloatAdd(&dest.x, inc.x);
		PxAtomicFloatAdd(&dest.y, inc.y);
		PxAtomicFloatAdd(&dest.z, inc.z);
	}
}

extern "C" __global__ void computeSmoothNormals(
	const PxClothParticle* particles,
	const PxU16* indices,
	Vertex* vertices,
	PxU32 numTris,
	PxU32 numParticles)
{
	// zero old normals
	for (PxU32 i=threadIdx.x; i < numParticles; i += blockDim.x)
		vertices[i].normal = PxVec3(0.0f);

	__syncthreads();

	for (PxU32 i=threadIdx.x; i < numTris; i += blockDim.x)
	{
		PxU16 a = indices[i*3];
		PxU16 b = indices[i*3+1];
		PxU16 c = indices[i*3+2];

		// calculate face normal
		PxVec3 e1 = particles[b].pos-particles[a].pos;
		PxVec3 e2 = particles[c].pos-particles[a].pos;
		PxVec3 n = e2.cross(e1);

		PxAtomicVec3Add(vertices[a].normal, n);
		PxAtomicVec3Add(vertices[b].normal, n);
		PxAtomicVec3Add(vertices[c].normal, n);
	}

	__syncthreads();

	// update vertex buffer
	for (PxU32 i=threadIdx.x; i < numParticles; i += blockDim.x)
	{
		vertices[i].position = particles[i].pos;
		vertices[i].normal = vertices[i].normal.getNormalized();
	}	
}

}